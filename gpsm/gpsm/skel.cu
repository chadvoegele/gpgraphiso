/* -*- mode: c++ -*- */

#include <hip/hip_runtime.h>
#include <cstdio>
#include <unistd.h>
#include <getopt.h>
#include <errno.h>

#include "gg.h"
#include "Timer.h"

#include "label.h"
#include "edgelist_graph.h"

int QUIET = 0;
char *DATA_GRAPH, *DATA_LABEL, *QUERY_GRAPH, *QUERY_LABEL, *OUTPUT;
unsigned long DISCOUNT_TIME_NS = 0;

extern int CUDA_DEVICE;

extern void gg_main(CSRGraphTy &, CSRGraphTy &, CSRGraphTy &, CSRGraphTy &, Shared<int>&, Shared<int>&);
extern const char *GGC_OPTIONS;

int load_graph_and_run_kernel(char *graph_file);
void output(CSRGraphTy &g, const char *output_file);

void kernel_sizing(CSRGraphTy & g, dim3 &blocks, dim3 &threads) {
  threads.x = 256;
  threads.y = threads.z = 1;

  blocks.x = ggc_get_nSM() * 8;
  blocks.y = blocks.z = 1;
}

void usage(int argc, char *argv[]) {
  fprintf(stderr, "usage: %s [-q] [-g gpunum] [-o output-file] data-graph.gr data-label.label query-graph.gr query-label.label\n", argv[0]);
}

void parse_args(int argc, char *argv[]) {
  int c;
  const char *opts = "g:qo:";

  while((c = getopt(argc, argv, opts)) != -1) {
    switch(c) {
    case 'q':
      QUIET = 1;
      break;
    case 'o':
      OUTPUT = optarg; //TODO: copy?
      break;
    case 'g':
      char *end;
      errno = 0;
      CUDA_DEVICE = strtol(optarg, &end, 10);
      if(errno != 0 || *end != '\0') {
        fprintf(stderr, "Invalid GPU device '%s'. An integer must be specified.\n", optarg);
        exit(EXIT_FAILURE);
      }
      break;
    case '?':
      usage(argc, argv);
      exit(EXIT_FAILURE);
    default:
      break;
    }
  }

  if (argc - optind == 4) {
    DATA_GRAPH = argv[optind];
    DATA_LABEL = argv[optind+1];
    QUERY_GRAPH = argv[optind+2];
    QUERY_LABEL = argv[optind+3];
  } else {
    usage(argc, argv);
    exit(EXIT_FAILURE);
  }
}

CSRGraphTy load_graph(char* graph) {
  CSRGraphTy g;

  char* ext = strrchr(graph, '.');
  if (!ext || ext == graph) {
    fprintf(stderr, "Unable to get graph file extension.\n");
    exit(EXIT_FAILURE);
  }

  if (!strcmp(".gr", ext)) {
    g.read(graph);

  } else if (!strcmp(".mtx", ext)) {
    gpgraphlib::EdgeListGraph elg = gpgraphlib::EdgeListGraph::fromMTXFile(graph);
    g.nnodes = elg.nnodes();
    g.nedges = elg.nedges();
    g.allocOnHost();
    elg.setCSR(g.row_start, g.edge_dst);

  } else {
    fprintf(stderr, "Unknown extension: %s\n. Supported: .gr, .mtx", ext);
    exit(EXIT_FAILURE);
  }

  return g;
}

int load_graph_and_run_kernel(char* data_graph, char* data_label, char* query_graph, char *query_label) {
  CSRGraphTy dg = load_graph(data_graph);
  CSRGraphTy dgg;
  dg.copy_to_gpu(dgg);
  gpgraphlib::LabelReader dlr = gpgraphlib::LabelReader::fromFilename(std::string(data_label));
  Shared<int> dprop = dg.nnodes;
  dlr.throwIfInvalid(dg.nnodes);
  int* dprop_cp = dprop.cpu_wr_ptr();
  dlr.setNodeProperties(dprop_cp);

  CSRGraphTy qg = load_graph(query_graph);
  CSRGraphTy qgg;
  qg.copy_to_gpu(qgg);
  gpgraphlib::LabelReader qlr = gpgraphlib::LabelReader::fromFilename(std::string(query_label));
  Shared<int> qprop = qg.nnodes;
  qlr.throwIfInvalid(qg.nnodes);
  int* qprop_cp = qprop.cpu_wr_ptr();
  qlr.setNodeProperties(qprop_cp);

  ggc::Timer k("gg_main");
  int *d;
  check_cuda(hipMalloc(&d, sizeof(int) * 1));

  k.start();
  gg_main(dg, dgg, qg, qgg, dprop, qprop);
  check_cuda(hipDeviceSynchronize());
  k.stop();
  k.print();
  fprintf(stderr, "Total time: %llu ms\n", k.duration_ms());
  fprintf(stderr, "Total time: %llu ns\n", k.duration());

  dgg.copy_to_cpu(dg);
  qgg.copy_to_cpu(qg);

  output(dg, OUTPUT);

  return EXIT_SUCCESS;
}

void output(CSRGraphTy &g, const char *output_file) {
  FILE *f;
  if(!output_file)
    return;
  if(strcmp(output_file, "-") == 0) {
    f = stdout;
  } else {
    f = fopen(output_file, "w");
  }
  check_fprintf(f, "%d\n", 1);
}

int main(int argc, char *argv[]) {
  CUDA_DEVICE = 0;  // default in case not set

  if(argc == 1) {
    usage(argc, argv);
    exit(1);
  }

  parse_args(argc, argv);
  printf("Data graph: %s, data labels: %s, query graph: %s, query_labels: %s\n", DATA_GRAPH, DATA_LABEL, QUERY_GRAPH, QUERY_LABEL);
  ggc_set_gpu_device(CUDA_DEVICE);
  int r = load_graph_and_run_kernel(DATA_GRAPH, DATA_LABEL, QUERY_GRAPH, QUERY_LABEL);
  return r;
}
