/* -*- mode: c++ -*- */

#include <hip/hip_runtime.h>
#include <cstdio>
#include <unistd.h>
#include <getopt.h>
#include <errno.h>

#include "gg.h"
#include "Timer.h"

extern void gg_main(CSRGraphTy &, CSRGraphTy &);
extern void output(CSRGraphTy &, const char *output_file);
extern const char *GGC_OPTIONS;

int QUIET = 0;
char *INPUT, *OUTPUT;
extern unsigned long DISCOUNT_TIME_NS;

unsigned long DISCOUNT_TIME_NS = 0;
int SKELAPP_RETVAL = 0;

extern int CUDA_DEVICE;

void kernel_sizing(CSRGraphTy & g, dim3 &blocks, dim3 &threads) {
  threads.x = 256;
  threads.y = threads.z = 1;

  blocks.x = ggc_get_nSM() * 8;
  blocks.y = blocks.z = 1;
}

int load_graph_and_run_kernel(char *graph_file) {
  CSRGraphTy g, gg;

  ggc::Timer k("gg_main");
  fprintf(stderr, "OPTIONS: %s\n", GGC_OPTIONS);
  g.read(graph_file);

  g.copy_to_gpu(gg);

  int *d;
  check_cuda(hipMalloc(&d, sizeof(int) * 1));

  k.start();
  gg_main(g, gg);
  check_cuda(hipDeviceSynchronize());
  k.stop();
  k.print();
  fprintf(stderr, "Total time: %llu ms\n", k.duration_ms());
  fprintf(stderr, "Total time: %llu ns\n", k.duration());

  if(DISCOUNT_TIME_NS > 0) {
    fprintf(stderr, "Total time (discounted): %llu ns\n", k.duration() - DISCOUNT_TIME_NS);
  }

  gg.copy_to_cpu(g);

  return SKELAPP_RETVAL;
}

void usage(int argc, char *argv[])
{
  fprintf(stderr, "usage: %s [-q] [-g gpunum] [-o output-file] input_graph.gr\n", argv[0]);
}

void parse_args(int argc, char *argv[])
{
  int c;
  const char *opts = "g:qo:";

  while((c = getopt(argc, argv, opts)) != -1) {
    switch(c)
      {
      case 'q':
	QUIET = 1;
	break;
      case 'o':
	OUTPUT = optarg; //TODO: copy?
	break;
      case 'g':
	char *end;
	errno = 0;
	CUDA_DEVICE = strtol(optarg, &end, 10);
	if(errno != 0 || *end != '\0') {
	  fprintf(stderr, "Invalid GPU device '%s'. An integer must be specified.\n", optarg);
	  exit(EXIT_FAILURE);
	}
	break;
      case '?':
	usage(argc, argv);
	exit(EXIT_FAILURE);
      default:
	break;
    }
  }

  INPUT = argv[optind];

  if(optind+1 != argc) {
    usage(argc, argv);
    exit(EXIT_FAILURE);
  }
}

int main(int argc, char *argv[]) {
  if(argc == 1) {
    usage(argc, argv);
    exit(1);
  }

  parse_args(argc, argv);
  ggc_set_gpu_device(CUDA_DEVICE);
  mgc = mgpu::CreateCudaDevice(CUDA_DEVICE);
  printf("Using GPU: %s\n", mgc->DeviceString().c_str());
  int r = load_graph_and_run_kernel(INPUT);
  return r;
}
