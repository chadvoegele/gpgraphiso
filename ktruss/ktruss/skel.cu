/* -*- mode: c++ -*- */

#include <hip/hip_runtime.h>
#include <cstdio>
#include <unistd.h>
#include <getopt.h>
#include <errno.h>

#include "gg.h"
#include "Timer.h"
#include "edgelist_graph.h"

extern void gg_main(CSRGraphTy &, CSRGraphTy &, unsigned);
extern void output(CSRGraphTy &, const char *output_file);
extern const char *GGC_OPTIONS;

int QUIET = 0;
char *INPUT, *OUTPUT;
unsigned ktruss_k = 3;
extern unsigned long DISCOUNT_TIME_NS;

unsigned long DISCOUNT_TIME_NS = 0;
int SKELAPP_RETVAL = 0;

extern int CUDA_DEVICE;

void kernel_sizing(CSRGraphTy & g, dim3 &blocks, dim3 &threads) {
  threads.x = 256;
  threads.y = threads.z = 1;

  blocks.x = ggc_get_nSM() * 8;
  blocks.y = blocks.z = 1;
}

CSRGraphTy load_graph(char* graph) {
  CSRGraphTy g;

  char* ext = strrchr(graph, '.');
  if (!ext || ext == graph) {
    fprintf(stderr, "Unable to get graph file extension.\n");
    exit(EXIT_FAILURE);
  }

  if (!strcmp(".gr", ext)) {
    g.read(graph);

  } else if (!strcmp(".mtx", ext)) {
    gpgraphlib::EdgeListGraph elg = gpgraphlib::EdgeListGraph::fromMTXFile(graph);
    g.nnodes = elg.nnodes();
    g.nedges = elg.nedges();
    g.allocOnHost();
    elg.setCSR(g.row_start, g.edge_dst);

  } else {
    fprintf(stderr, "Unknown extension: %s\n. Supported: .gr, .mtx", ext);
    exit(EXIT_FAILURE);
  }

  return g;
}

int load_graph_and_run_kernel(char *graph_file) {
  ggc::Timer k("gg_main");
  fprintf(stderr, "OPTIONS: %s\n", GGC_OPTIONS);

  CSRGraphTy g = load_graph(graph_file);
  CSRGraphTy gg;
  g.copy_to_gpu(gg);

  int *d;
  check_cuda(hipMalloc(&d, sizeof(int) * 1));

  k.start();
  gg_main(g, gg, ktruss_k);
  check_cuda(hipDeviceSynchronize());
  k.stop();
  k.print();
  fprintf(stderr, "Total time: %llu ms\n", k.duration_ms());
  fprintf(stderr, "Total time: %llu ns\n", k.duration());

  if(DISCOUNT_TIME_NS > 0) {
    fprintf(stderr, "Total time (discounted): %llu ns\n", k.duration() - DISCOUNT_TIME_NS);
  }

  gg.copy_to_cpu(g);

  return SKELAPP_RETVAL;
}

void usage(int argc, char *argv[])
{
  fprintf(stderr, "usage: %s [-q] [-g gpunum] [-o output-file] [-k #] input_graph.gr\n", argv[0]);
}

void parse_args(int argc, char *argv[])
{
  int c;
  const char *opts = "g:qo:k:";

  while((c = getopt(argc, argv, opts)) != -1) {
    switch(c)
      {
      case 'q':
	QUIET = 1;
	break;
      case 'o':
	OUTPUT = optarg; //TODO: copy?
	break;
      case 'k':
	ktruss_k = atoi(optarg); //TODO: copy?
	break;
      case 'g':
	char *end;
	errno = 0;
	CUDA_DEVICE = strtol(optarg, &end, 10);
	if(errno != 0 || *end != '\0') {
	  fprintf(stderr, "Invalid GPU device '%s'. An integer must be specified.\n", optarg);
	  exit(EXIT_FAILURE);
	}
	break;
      case '?':
	usage(argc, argv);
	exit(EXIT_FAILURE);
      default:
	break;
    }
  }

  INPUT = argv[optind];

  if(optind+1 != argc) {
    usage(argc, argv);
    exit(EXIT_FAILURE);
  }
}

int main(int argc, char *argv[]) {
  if(argc == 1) {
    usage(argc, argv);
    exit(1);
  }

  parse_args(argc, argv);
  ggc_set_gpu_device(CUDA_DEVICE);
  mgc = mgpu::CreateCudaDevice(CUDA_DEVICE);
  printf("Using GPU: %s\n", mgc->DeviceString().c_str());
  int r = load_graph_and_run_kernel(INPUT);
  return r;
}
