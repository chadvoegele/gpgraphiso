/* -*- mode: c++ -*- */

#include <hip/hip_runtime.h>
#include <cstdio>
#include <unistd.h>
#include <getopt.h>
#include <errno.h>

#include "gg.h"
#include "Timer.h"
#include "edgelist_graph.h"

extern void gg_main(CSRGraphTy &, CSRGraphTy &, unsigned, char*, FILE*);

FILE* OUTF = 0;
int QUIET = 0;
char *INPUT, *OUTPUT;
const char* OUTPUTKIND_DEFAULT = "counts|edges";
char* OUTPUTKIND = (char*)OUTPUTKIND_DEFAULT;
unsigned ktruss_k = 3;

int SKELAPP_RETVAL = 0;
mgpu::ContextPtr mgc;

int CUDA_DEVICE;

void kernel_sizing(CSRGraphTy & g, dim3 &blocks, dim3 &threads) {
  threads.x = 256;
  threads.y = threads.z = 1;

  blocks.x = ggc_get_nSM() * 8;
  blocks.y = blocks.z = 1;
}

CSRGraphTy load_graph(char* graph) {
  CSRGraphTy g;

  char* ext = strrchr(graph, '.');
  if (!ext || ext == graph) {
    fprintf(stderr, "Unable to get graph file extension.\n");
    exit(EXIT_FAILURE);
  }

  if (!strcmp(".gr", ext)) {
    g.read(graph);

  } else if (!strcmp(".mtx", ext)) {
    gpgraphlib::EdgeListGraph elg = gpgraphlib::EdgeListGraph::fromMTXFile(graph);
    g.nnodes = elg.nnodes();
    g.nedges = elg.nedges();
    g.allocOnHost();
    elg.setCSR(g.row_start, g.edge_dst);

  } else {
    fprintf(stderr, "Unknown extension: %s\n. Supported: .gr, .mtx", ext);
    exit(EXIT_FAILURE);
  }

  return g;
}

int load_graph_and_run_kernel(char *graph_file) {
  CSRGraphTy g = load_graph(graph_file);
  CSRGraphTy gg;

  int *d;
  check_cuda(hipMalloc(&d, sizeof(int) * 1));

  g.copy_to_gpu(gg);
  gg_main(g, gg, ktruss_k, OUTPUTKIND, OUTF);
  check_cuda(hipDeviceSynchronize());

  return SKELAPP_RETVAL;
}

void usage(int argc, char *argv[])
{
  fprintf(stderr, "usage: %s [-q] [-g gpunum] [-o output-file] [-k #] [-p output-kind] input_graph.gr\n", argv[0]);
  fprintf(stderr, "       output-kind=\"edges|counts\"\n", argv[0]);
}

void parse_args(int argc, char *argv[])
{
  int c;
  const char *opts = "g:qo:k:p:";

  while((c = getopt(argc, argv, opts)) != -1) {
    switch(c)
      {
      case 'q':
	QUIET = 1;
	break;
      case 'o':
	OUTPUT = optarg; //TODO: copy?
	if (!OUTF) {
	  if (strcmp(OUTPUT, "-") == 0) {
	    OUTF = stdout;
	  } else {
	    OUTF = fopen(OUTPUT, "w");
	  }
	}
	break;
      case 'k':
	ktruss_k = atoi(optarg); //TODO: copy?
	break;
      case 'g':
	char *end;
	errno = 0;
	CUDA_DEVICE = strtol(optarg, &end, 10);
	if(errno != 0 || *end != '\0') {
	  fprintf(stderr, "Invalid GPU device '%s'. An integer must be specified.\n", optarg);
	  exit(EXIT_FAILURE);
	}
	break;
      case 'p':
	OUTPUTKIND = optarg;
	break;
      case '?':
	usage(argc, argv);
	exit(EXIT_FAILURE);
      default:
	break;
    }
  }

  INPUT = argv[optind];

  if(optind+1 != argc) {
    usage(argc, argv);
    exit(EXIT_FAILURE);
  }
}

int main(int argc, char *argv[]) {
  if(argc == 1) {
    usage(argc, argv);
    exit(1);
  }

  parse_args(argc, argv);
  ggc_set_gpu_device(CUDA_DEVICE);
  mgc = mgpu::CreateCudaDevice(CUDA_DEVICE);
  printf("Using GPU: %s\n", mgc->DeviceString().c_str());
  int r = load_graph_and_run_kernel(INPUT);
  return r;
}
