/* -*- mode: c++ -*- */

#include <hip/hip_runtime.h>
#include <cstdio>
#include <unistd.h>
#include <getopt.h>
#include <errno.h>
#include <hip/hip_runtime_api.h>

#include "gg.h"
#include "Timer.h"
#include "edgelist_graph.h"
#include "ktruss.h"

extern void gg_main(CSRGraphTy &, CSRGraphTy &, unsigned, Shared<unsigned char>&, unsigned&, unsigned&);

FILE* OUTF = 0;
int QUIET = 0;
char *INPUT, *OUTPUT;
const char* OUTPUTKIND_DEFAULT = "counts|edges";
char* OUTPUTKIND = (char*)OUTPUTKIND_DEFAULT;
unsigned ktruss_k = 3;

int SKELAPP_RETVAL = 0;
mgpu::ContextPtr mgc;

int CUDA_DEVICE;

void kernel_sizing(CSRGraphTy & g, dim3 &blocks, dim3 &threads) {
  threads.x = 256;
  threads.y = threads.z = 1;

  blocks.x = ggc_get_nSM() * 8;
  blocks.y = blocks.z = 1;
}

CSRGraphTy load_graph(char* graph) {
  CSRGraphTy g;

  char* ext = strrchr(graph, '.');
  if (!ext || ext == graph) {
    fprintf(stderr, "Unable to get graph file extension.\n");
    exit(EXIT_FAILURE);
  }

  if (!strcmp(".gr", ext)) {
    g.read(graph);

  } else if (!strcmp(".mtx", ext)) {
    gpgraphlib::EdgeListGraph elg = gpgraphlib::EdgeListGraph::fromMTXFile(graph);
    g.nnodes = elg.nnodes();
    g.nedges = elg.nedges();
    g.allocOnHost();
    elg.setCSR(g.row_start, g.edge_dst);

  } else {
    fprintf(stderr, "Unknown extension: %s\n. Supported: .gr, .mtx", ext);
    exit(EXIT_FAILURE);
  }

  return g;
}

int load_graph_and_run_kernel(char *graph_file) {
  CSRGraphTy g = load_graph(graph_file);
  CSRGraphTy gg;

  int *d;
  check_cuda(hipMalloc(&d, sizeof(int) * 1));

  Shared<unsigned char> eremoved (g.nedges);

  g.copy_to_gpu(gg);
  ggc::Timer timer("gg_main");
  unsigned n_ktruss_nodes, n_ktruss_edges;

  timer.start();

  gg_main(g, gg, ktruss_k, eremoved, n_ktruss_nodes, n_ktruss_edges);
  check_cuda(hipDeviceSynchronize());
  timer.stop();

  fprintf(stderr, "Total time: %llu ms\n", timer.duration_ms());
  fprintf(stderr, "Total time: %llu ns\n", timer.duration());

  if(OUTPUT) {
    gg.copy_to_cpu(g);
    output(g, OUTPUTKIND, OUTF, n_ktruss_nodes, n_ktruss_edges, eremoved.cpu_rd_ptr());
  }

  return SKELAPP_RETVAL;
}

void usage(int argc, char *argv[])
{
  fprintf(stderr, "usage: %s [-q] [-g gpunum] [-o output-file] [-k #] [-p output-kind] input_graph.gr\n", argv[0]);
  fprintf(stderr, "       output-kind=\"edges|counts\"\n", argv[0]);
}

void parse_args(int argc, char *argv[])
{
  int c;
  const char *opts = "g:qo:k:p:";

  while((c = getopt(argc, argv, opts)) != -1) {
    switch(c)
      {
      case 'q':
	QUIET = 1;
	break;
      case 'o':
	OUTPUT = optarg; //TODO: copy?
	if (!OUTF) {
	  if (strcmp(OUTPUT, "-") == 0) {
	    OUTF = stdout;
	  } else {
	    OUTF = fopen(OUTPUT, "w");
	  }
	}
	break;
      case 'k':
	ktruss_k = atoi(optarg); //TODO: copy?
	break;
      case 'g':
	char *end;
	errno = 0;
	CUDA_DEVICE = strtol(optarg, &end, 10);
	if(errno != 0 || *end != '\0') {
	  fprintf(stderr, "Invalid GPU device '%s'. An integer must be specified.\n", optarg);
	  exit(EXIT_FAILURE);
	}
	break;
      case 'p':
	OUTPUTKIND = optarg;
	break;
      case '?':
	usage(argc, argv);
	exit(EXIT_FAILURE);
      default:
	break;
    }
  }

  INPUT = argv[optind];

  if(optind+1 != argc) {
    usage(argc, argv);
    exit(EXIT_FAILURE);
  }
}

void dump_memory_info(const char *s) {
  size_t total, free;

  if(hipMemGetInfo(&free, &total) == hipSuccess) {
    fprintf(stderr, "INSTR gpu_memory_total_%s %zu\n", s, total);
    fprintf(stderr, "INSTR gpu_memory_free_%s %zu\n", s, free);
  }
}

int main(int argc, char *argv[]) {
  if(argc == 1) {
    usage(argc, argv);
    exit(1);
  }

  parse_args(argc, argv);
  dump_memory_info("start");
  ggc_set_gpu_device(CUDA_DEVICE);
  mgc = mgpu::CreateCudaDevice(CUDA_DEVICE);
  printf("Using GPU: %s\n", mgc->DeviceString().c_str());
  int r = load_graph_and_run_kernel(INPUT);
  hipProfilerStop();
  return r;
}
